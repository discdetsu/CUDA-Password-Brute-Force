
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <ctime>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

const string ALPHABET_SET = "0123456789!@#$%^&*abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ";

// device utility functions
__device__
int cuda_strcmp(char* str1, char* str2, int length);
__device__
void cuda_strcpy(char *dest, char *src);
__device__
int cuda_strlen(char *string)
{
    int count = 0;
	while (string[count] != '\0') {
		++count;
	}
	return count;
}
__device__
bool check(char *pass, char c1, char c2, char c3){

    if (c1 == pass[0] && c2 == pass[1] && c3 == pass[2])
        return true;

    return false;
}

//kernel
__global__
void random_password(char* pass, char* alphabet_set, unsigned int seed)
{

    extern __shared__ char alphabet[];

    char test[10];
    int passLen = cuda_strlen(pass);
    int alphabet_length = cuda_strlen(alphabet_set);

    for (int i = 0; i<alphabet_length; i++)
        alphabet[i] = alphabet_set[i];

    hiprandState_t state;
    int rand;
    hiprand_init(seed,0,0,&state);
    

    for(int i = 0; i < passLen; i++){
        rand = hiprand(&state) % alphabet_length;
        test[i] = alphabet[rand];
    }
    test[passLen] ='\0';

    for(int i = 0; i < passLen; i++){
        printf("%c", test[i]);
    }



}
__global__
void bruteforce(char* pass)
{
    
        printf("Block ID: %d, Thread ID: %d, Block Dimension: %d\n", blockIdx.x, threadIdx.x, blockDim.x);
        printf("Index: %d\n",  threadIdx.x + blockDim.x *  blockIdx.x);
  
        // printf("%c%c%c\n", (char)c1, (char)blockIdx.x, (char)threadIdx.x + 31);
        
    
}

//driver code
int main()
{
    string pass = "abc";
    char *d_pass, *d_alphabet;
    hipMalloc((void**)&d_pass, sizeof(char)*pass.length() + 1);
    hipMalloc((void**)&d_alphabet, sizeof(char)*ALPHABET_SET.length() + 1);
    hipMemcpy(d_pass, pass.c_str(), sizeof(char)*pass.length() + 1, hipMemcpyHostToDevice);
    hipMemcpy(d_alphabet, ALPHABET_SET.c_str(), sizeof(char)*ALPHABET_SET.length() + 1, hipMemcpyHostToDevice);
    random_password<<<1,1, sizeof(char) * ALPHABET_SET.length()>>>(d_pass, d_alphabet, time(NULL));

    hipFree(d_alphabet);
    hipFree(d_pass);
    return -1;
}