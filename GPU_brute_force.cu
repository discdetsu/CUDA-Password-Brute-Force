
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <iomanip>
#include <ctime>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

const string ALPHABET_SET = "0123456789abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ";

// device utility functions
__device__
int c_strlen(char *string)
{
    int count = 0;
	while (string[count] != '\0') {
		++count;
	}
	return count;
}

//kernel
// __global__
// void random_password(char* pass, char* alphabet_set, unsigned int seed)
// {

//     extern __shared__ char alphabet[];

//     char test[10];
//     int passLen = c_strlen(pass);
//     int a_l = c_strlen(alphabet_set);

//     for (int i = 0; i<a_l; i++)
//         alphabet[i] = alphabet_set[i];

//     // int digit[8];
//     // digit[0] = blockIdx.x;

//     printf("Block ID: %d\n", blockIdx.x);
//     printf("Thread ID: %d\n", threadIdx.x);

//     // for radom index from alphabet set range
//     curandState_t state;
//     int rand;
//     curand_init(seed,0,0,&state);

//     for(int i = 0; i < passLen; i++){
//         rand = curand(&state) % a_l;
//         test[i] = alphabet[rand];
//     }
//     test[passLen] ='\0';

//     for(int i = 0; i < passLen; i++){
//         printf("%c", test[i]);
//     }
   
// }




__global__
void bruteforce(char* pass, char* alphabet_set)
{
    int passLen = c_strlen(pass);
    int a_l = c_strlen(alphabet_set); // Alphabet length
   

    if (passLen == 1){ //call kernel by <<<1,1>>
        for (int i = 0; i < a_l; i++){
            printf("%c\n",alphabet_set[i]); 
        }
    }
    else if (passLen == 2){ //call kernel by <<<1,len>>>
        for (int i = 0; i < a_l; i++){
            printf("%c%c\n",alphabet_set[i], 
                            alphabet_set[threadIdx.x]); 
        }
    }
    else if (passLen == 3){ //call kernel by <<<len,len>>>
        for (int i = 0; i < a_l; i++){
            printf("%c%c%c\n", alphabet_set[i], 
                            alphabet_set[threadIdx.x],
                            alphabet_set[(int)(blockIdx.x % a_l)]);
        }
    }
    else if (passLen == 4){ //call kernel by <<<len^2,len>>>
        for (int i = 0; i < a_l; i++){
            printf("%c%c%c%c\n", alphabet_set[i], 
                                alphabet_set[threadIdx.x], 
                                alphabet_set[(int)(blockIdx.x % a_l)],
                                alphabet_set[(int)((blockIdx.x / a_l ) % a_l)]);
                
        }
    }    
    else if (passLen == 5){ //call kernel by <<<len^3,len>>>
        for (int i = 0; i < a_l; i++){
            printf("%c%c%c%c%c\n", alphabet_set[i], 
                                alphabet_set[threadIdx.x], 
                                alphabet_set[(int)(blockIdx.x % a_l)],
                                alphabet_set[(int)((blockIdx.x / a_l ) % a_l)],
                                alphabet_set[(int)((blockIdx.x / (a_l*a_l)) % a_l)]);
                
        }
    }
    else if (passLen == 6){ //call kernel by <<<len^4,len>>>
        for (int i = 0; i < a_l; i++){
            printf("%c%c%c%c%c%c\n", alphabet_set[i], 
                                alphabet_set[threadIdx.x], 
                                alphabet_set[(int)(blockIdx.x % a_l)],
                                alphabet_set[(int)((blockIdx.x / a_l ) % a_l)],
                                alphabet_set[(int)((blockIdx.x / (a_l*a_l)) % a_l)],
                                alphabet_set[(int)((blockIdx.x / (a_l*a_l*a_l)) % a_l)]);
                
        }
    }
    else if (passLen == 7){ //call kernel by <<<len^5,len>>>
        for (int i = 0; i < a_l; i++){
            printf("%c%c%c%c%c%c%c\n", alphabet_set[i], 
                                alphabet_set[threadIdx.x], 
                                alphabet_set[(int)(blockIdx.x % a_l)],
                                alphabet_set[(int)((blockIdx.x / a_l ) % a_l)],
                                alphabet_set[(int)((blockIdx.x / (a_l*a_l)) % a_l)],
                                alphabet_set[(int)((blockIdx.x / (a_l*a_l*a_l)) % a_l)],
                                alphabet_set[(int)((blockIdx.x / (a_l*a_l*a_l*a_l)) % a_l)]);
                
        }
    }
    else if (passLen == 8){ //call kernel by <<<len^6,len>>>
        for (int i = 0; i < a_l; i++){
            printf("%c%c%c%c%c%c%c%c\n", alphabet_set[i], 
                                alphabet_set[threadIdx.x], 
                                alphabet_set[(int)(blockIdx.x % a_l)],
                                alphabet_set[(int)((blockIdx.x / a_l ) % a_l)],
                                alphabet_set[(int)((blockIdx.x / (a_l*a_l)) % a_l)],
                                alphabet_set[(int)((blockIdx.x / (a_l*a_l*a_l)) % a_l)],
                                alphabet_set[(int)((blockIdx.x / (a_l*a_l*a_l*a_l)) % a_l)],
                                alphabet_set[(int)((blockIdx.x / (a_l*a_l*a_l*a_l*a_l)) % a_l)]);
                
        }
    }
        
    
}



//driver code
int main()
{

    hipEvent_t start, stop; //timer
    float ms;
    hipEventCreate(&start);
	hipEventCreate(&stop);

    string password;

    cout << "Please enter password: ";
    cin >> password;

    char* d_pass;
    char* d_alphabet_set;

    hipMalloc((void**)&d_pass, sizeof(char)*password.length() + 1);
    hipMalloc((void**)&d_alphabet_set, sizeof(char)*ALPHABET_SET.length() + 1);
   
    hipMemcpy(d_pass, password.c_str(), sizeof(char)*password.length() + 1, hipMemcpyHostToDevice);
    hipMemcpy(d_alphabet_set, ALPHABET_SET.c_str(), sizeof(char)*ALPHABET_SET.length() + 1, hipMemcpyHostToDevice);
   


    int blocksPerGrid;
    int threadsPerBlock;
    if (password.length() == 1){
        blocksPerGrid = 1;
        threadsPerBlock = 1;
    }
    else if (password.length() == 2){
        threadsPerBlock = ALPHABET_SET.length();
        blocksPerGrid = 1;
    }
    else if (password.length() == 3){
        threadsPerBlock = ALPHABET_SET.length();
        blocksPerGrid = ALPHABET_SET.length();
    }
    else {
        threadsPerBlock = ALPHABET_SET.length();
        blocksPerGrid = (int)std::pow((float)ALPHABET_SET.length(), password.length() - 2);
    }

    cout << blocksPerGrid << endl;

    hipEventRecord(start);
    bruteforce<<<blocksPerGrid,threadsPerBlock>>>(d_pass, d_alphabet_set);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    cout << "All combination with password length: " << password.length() << " in: " << ms << " milliseconds." << endl;

    return -1;
}