
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>

using namespace std;

const string ALPHABET_SET = "A-Za-z";

// device utility functions
__device__
int cuda_strcmp(char* str1, char* str2, int length);
__device__
void cuda_strcpy(char *dest, char *src);
__device__
int cuda_strlen(char *string);

//kernel
__global__
void bruteforce();

//driver code
int main()
{
    return -1;
}