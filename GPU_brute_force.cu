
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <iomanip>
#include <ctime>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

const string ALPHABET_SET = "0123456789abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ";

// device utility functions
__device__
int c_strcmp(char* str1, char* str2, int length)
{
    int flag = 0;

	for (int i = 0; i<length; i++) {
		if (str1[i] != str2[i]) {
			flag = 1;
			break;
		}
	}

	return flag;
}

__device__
void c_strcpy(char *dest, char *src)
{
    int i = 0;
	do {
		dest[i] = src[i];
	} while (src[++i] != '\0');
}

__device__
int c_strlen(char *string)
{
    int count = 0;
	while (string[count] != '\0') {
		++count;
	}
	return count;
}

//kernel
__global__
void random_password(char* pass, char* alphabet_set, unsigned int seed)
{

    extern __shared__ char alphabet[];

    char test[10];
    int passLen = c_strlen(pass);
    int a_l = c_strlen(alphabet_set);

    for (int i = 0; i<a_l; i++)
        alphabet[i] = alphabet_set[i];

    // int digit[8];
    // digit[0] = blockIdx.x;

    printf("Block ID: %d\n", blockIdx.x);
    printf("Thread ID: %d\n", threadIdx.x);

    // for radom index from alphabet set range
    hiprandState_t state;
    int rand;
    hiprand_init(seed,0,0,&state);

    for(int i = 0; i < passLen; i++){
        rand = hiprand(&state) % a_l;
        test[i] = alphabet[rand];
    }
    test[passLen] ='\0';

    for(int i = 0; i < passLen; i++){
        printf("%c", test[i]);
    }
   
}
__global__
void bruteforce(char* pass, char* alphabet_set, char* generated_pass)
{
    int passLen = c_strlen(pass);
    int a_l = c_strlen(alphabet_set); // Alphabet length
   

    if (passLen == 1){ //call kernel by <<<1,1>>
        for (int i = 0; i < a_l; i++){
            printf("%c\n",alphabet_set[i]); 
        }
    }
    else if (passLen == 2){ //call kernel by <<<1,len>>>
        for (int i = 0; i < a_l; i++){
            printf("%c%c\n",alphabet_set[i], 
                            alphabet_set[threadIdx.x]); 
        }
    }
    else if (passLen == 3){ //call kernel by <<<len,len>>>
        for (int i = 0; i < a_l; i++){
            printf("%c%c%c\n", alphabet_set[i], 
                            alphabet_set[threadIdx.x],
                            alphabet_set[(int)(blockIdx.x % a_l)]);
        }
    }
    else if (passLen == 4){ //call kernel by <<<len^4,len>>>
        for (int i = 0; i < a_l; i++){
            printf("%c%c%c%c\n", alphabet_set[i], 
                                alphabet_set[threadIdx.x], 
                                alphabet_set[(int)(blockIdx.x % a_l)],
                                alphabet_set[(int)((blockIdx.x / a_l ) % a_l)]);
                
        }
    }    
    else if (passLen == 5){ //call kernel by <<<len^5,len>>>
        for (int i = 0; i < a_l; i++){
            printf("%c%c%c%c%c\n", alphabet_set[i], 
                                alphabet_set[threadIdx.x], 
                                alphabet_set[(int)(blockIdx.x % a_l)],
                                alphabet_set[(int)((blockIdx.x / a_l ) % a_l)],
                                alphabet_set[(int)((blockIdx.x / (a_l*a_l)) % a_l)]);
                
        }
    }
    else if (passLen == 6){ //call kernel by <<<len^6,len>>>
        for (int i = 0; i < a_l; i++){
            printf("%c%c%c%c%c%c\n", alphabet_set[i], 
                                alphabet_set[threadIdx.x], 
                                alphabet_set[(int)(blockIdx.x % a_l)],
                                alphabet_set[(int)((blockIdx.x / a_l ) % a_l)],
                                alphabet_set[(int)((blockIdx.x / (a_l*a_l)) % a_l)],
                                alphabet_set[(int)((blockIdx.x / (a_l*a_l*a_l)) % a_l)]);
                
        }
    }
    else if (passLen == 7){ //call kernel by <<<len^7,len>>>
        for (int i = 0; i < a_l; i++){
            printf("%c%c%c%c%c%c%c\n", alphabet_set[i], 
                                alphabet_set[threadIdx.x], 
                                alphabet_set[(int)(blockIdx.x % a_l)],
                                alphabet_set[(int)((blockIdx.x / a_l ) % a_l)],
                                alphabet_set[(int)((blockIdx.x / (a_l*a_l)) % a_l)],
                                alphabet_set[(int)((blockIdx.x / (a_l*a_l*a_l)) % a_l)],
                                alphabet_set[(int)((blockIdx.x / (a_l*a_l*a_l*a_l)) % a_l)]);
                
        }
    }
    else if (passLen == 8){ //call kernel by <<<len^8,len>>>
        for (int i = 0; i < a_l; i++){
            printf("%c%c%c%c%c%c%c%c\n", alphabet_set[i], 
                                alphabet_set[threadIdx.x], 
                                alphabet_set[(int)(blockIdx.x % a_l)],
                                alphabet_set[(int)((blockIdx.x / a_l ) % a_l)],
                                alphabet_set[(int)((blockIdx.x / (a_l*a_l)) % a_l)],
                                alphabet_set[(int)((blockIdx.x / (a_l*a_l*a_l)) % a_l)],
                                alphabet_set[(int)((blockIdx.x / (a_l*a_l*a_l*a_l)) % a_l)],
                                alphabet_set[(int)((blockIdx.x / (a_l*a_l*a_l*a_l*a_l)) % a_l)]);
                
        }
    }
        
    
}



//driver code
int main()
{

    string password;

    cout << "Please enter password to crack: ";
    cin >> password;

    char* d_pass;
    char* d_alphabet_set;
    char* d_generated_pass;
    char* result = (char*)malloc(sizeof(char)*password.length() + 1);

    hipMalloc((void**)&d_pass, sizeof(char)*password.length() + 1);
    hipMalloc((void**)&d_alphabet_set, sizeof(char)*ALPHABET_SET.length() + 1);
    hipMalloc((void**)&d_generated_pass, sizeof(char)*password.length() + 1);
    hipMemcpy(d_pass, password.c_str(), sizeof(char)*password.length() + 1, hipMemcpyHostToDevice);
    hipMemcpy(d_alphabet_set, ALPHABET_SET.c_str(), sizeof(char)*ALPHABET_SET.length() + 1, hipMemcpyHostToDevice);
    hipMemcpy(d_generated_pass, result, sizeof(char)*password.length() + 1, hipMemcpyHostToDevice);

    // while (true)
    // {
        int a_l = ALPHABET_SET.length();
        int p_l = password.length();
             
        bruteforce<<<1,1>>>(d_pass, d_alphabet_set, d_generated_pass);

        hipMemcpy(result, d_generated_pass, sizeof(char)*password.length() + 1, hipMemcpyDeviceToHost);

        free(result);

    // }

    return -1;
}