
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <ctime>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

const string ALPHABET_SET = "0123456789!@#$%^&*abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ";

// device utility functions
__device__
int strcmp(char* str1, char* str2, int length);

__device__
void strcpy(char *dest, char *src)
{
    int i = 0;
	do {
		dest[i] = src[i];
	} while (src[++i] != '\0');
}

__device__
int strlen(char *string)
{
    int count = 0;
	while (string[count] != '\0') {
		++count;
	}
	return count;
}

//kernel
__global__
void random_password(char* pass, char* alphabet_set, unsigned int seed)
{

    extern __shared__ char alphabet[];

    char test[10];
    int passLen = strlen(pass);
    int alphabet_length = strlen(alphabet_set);

    for (int i = 0; i<alphabet_length; i++)
        alphabet[i] = alphabet_set[i];

    // int digit[8];
    // digit[0] = blockIdx.x;

    printf("Block ID: %d\n", blockIdx.x);
    printf("Thread ID: %d\n", threadIdx.x);

    // for radom index from alphabet set range
    hiprandState_t state;
    int rand;
    hiprand_init(seed,0,0,&state);

    for(int i = 0; i < passLen; i++){
        rand = hiprand(&state) % alphabet_length;
        test[i] = alphabet[rand];
    }
    test[passLen] ='\0';

    for(int i = 0; i < passLen; i++){
        printf("%c", test[i]);
    }
   
}
__global__
void bruteforce()
{
    char pass[] = "caaaaa";
    char alphabet_set[] = "abc";
    int passLen = strlen(pass);
    int alphabet_length = strlen(alphabet_set);


    if (passLen == 1){
        for (int i = 0; i < alphabet_length; i++){
            printf("%c\n",alphabet_set[i]); //call kernel <<<1,1>>
        }
    }
    else if (passLen == 2){
        for (int i = 0; i < alphabet_length; i++){
            printf("%c%c\n",alphabet_set[i], 
                            alphabet_set[threadIdx.x]); //call kernel <<<1,3>>>
        }
    }
    else if (passLen == 3){
        for (int i = 0; i < alphabet_length; i++){
            printf("%c%c%c\n",alphabet_set[i], 
                            alphabet_set[threadIdx.x],
                            alphabet_set[(int)(blockIdx.x % alphabet_length)]);
        }
    }
    else if (passLen == 4){
        for (int i = 0; i < alphabet_length; i++){
            printf("%c%c%c%c\n", alphabet_set[i], 
                                alphabet_set[threadIdx.x], 
                                alphabet_set[(int)(blockIdx.x % alphabet_length)],
                                alphabet_set[(int)((blockIdx.x / alphabet_length ) % alphabet_length)]);
                
        }
    }    
    else if (passLen == 5){
        for (int i = 0; i < alphabet_length; i++){
            printf("%c%c%c%c%c\n", alphabet_set[i], 
                                alphabet_set[threadIdx.x], 
                                alphabet_set[(int)(blockIdx.x % alphabet_length)],
                                alphabet_set[(int)((blockIdx.x / alphabet_length ) % alphabet_length)],
                                alphabet_set[(int)((blockIdx.x / (alphabet_length*alphabet_length)) % alphabet_length)]);
                
        }
    }
    else if (passLen == 6){
        for (int i = 0; i < alphabet_length; i++){
            printf("%c%c%c%c%c%c\n", alphabet_set[i], 
                                alphabet_set[threadIdx.x], 
                                alphabet_set[(int)(blockIdx.x % alphabet_length)],
                                alphabet_set[(int)((blockIdx.x / alphabet_length ) % alphabet_length)],
                                alphabet_set[(int)((blockIdx.x / (alphabet_length*alphabet_length)) % alphabet_length)],
                                alphabet_set[(int)((blockIdx.x / (alphabet_length*alphabet_length*alphabet_length)) % alphabet_length)]);
                
        }
    }
        
    
}

//driver code
int main()
{
    // string pass = "abc";
    // char *d_pass, *d_alphabet;
    // cudaMalloc((void**)&d_pass, sizeof(char)*pass.length() + 1);
    // cudaMalloc((void**)&d_alphabet, sizeof(char)*ALPHABET_SET.length() + 1);
    // cudaMemcpy(d_pass, pass.c_str(), sizeof(char)*pass.length() + 1, cudaMemcpyHostToDevice);
    // cudaMemcpy(d_alphabet, ALPHABET_SET.c_str(), sizeof(char)*ALPHABET_SET.length() + 1, cudaMemcpyHostToDevice);
    // random_password<<<1,1, sizeof(char) * ALPHABET_SET.length()>>>(d_pass, d_alphabet, time(NULL));
    
    // cudaFree(d_alphabet);
    // cudaFree(d_pass);


    bruteforce<<<3*3*3*3*3*3,3>>>();

    return -1;
}